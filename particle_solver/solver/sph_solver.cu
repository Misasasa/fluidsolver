#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <thrust\device_vector.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>


#include "hip/hip_runtime_api.h"
#include "helper_string.h"
#include "sph_kernel_impl.cuh"

#include "sph_solver.cuh"

namespace sph{

void copyDeviceBuffer() {
	hipMemcpyToSymbol(HIP_SYMBOL(dParam), &hParam, sizeof(SimParam_SPH));
}

void fetchDeviceBuffer() {
	hipMemcpyFromSymbol(&hParam, HIP_SYMBOL(dParam), sizeof(SimParam_SPH));
}





void calcHash(SimData_SPH data,	int numParticles) {

	getLastCudaError("Kernel execution failed:before calc hash");
	uint numBlocks, numThreads;
	computeGridSize(numParticles, 256, numBlocks, numThreads);

	calcHashD << <numBlocks, numThreads >> > (data.particleHash,
		data.particleIndex,
		data.pos,
		numParticles);
	hipDeviceSynchronize();
	getLastCudaError("Kernel execution failed: calc hash");
}



void sortParticle(SimData_SPH data,	int pnum) {
	thrust::sort_by_key(
		thrust::device_ptr<int>(data.particleHash),
		thrust::device_ptr<int>(data.particleHash + pnum),
		thrust::device_ptr<int>(data.particleIndex)
	);

}



void reorderDataAndFindCellStart(
	SimData_SPH data, 
	int numParticles,
	int numGridCells
) {
	uint numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);

	hipMemset(data.gridCellStart, 0xffffffff, numGridCells * sizeof(uint));

	//shared memory size
	uint smemSize = sizeof(uint)*(numThreads + 1);
	
	reorderDataAndFindCellStartD << < numBlocks, numThreads, smemSize >> >(
		data,
		numParticles);
	
	getLastCudaError("Kernel execution failed: reorder data");

}



void applyXSPH(SimData_SPH data, int numParticles) {


}



void computePressure(SimData_SPH data, int numP) {
	uint numThreads, numBlocks;
	computeGridSize(numP, 256, numBlocks, numThreads);
	
	computeP <<< numBlocks, numThreads>>>(data, numP);
}

void computeForce(SimData_SPH data, int numP) {
	uint numThreads, numBlocks;
	computeGridSize(numP, 256, numBlocks, numThreads);

	computeF <<< numBlocks, numThreads>>>(data, numP);
}

void advect(SimData_SPH data, int numP) {
	uint numThreads, numBlocks;
	computeGridSize(numP, 256, numBlocks, numThreads);

	advectAndCollision <<< numBlocks, numThreads>>> (data, numP);
}



};